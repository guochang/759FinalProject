#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>

#include "hipblas.h"

#define N 16
#define n 4

void check(int *puzzle_sol){
	hipError_t cudaStat;
	hipblasStatus_t stat;
	hipblasHandle_t handle;

	int i, j;
	float* A;//store the puzzle solution
	float* A_sub;//
	float* x;//multiply with the matrix to get the sum
	float* y;//store the sum->difference
	float checksum[16] = { 136, 136, 136, 136, 136, 136, 136, 136, 136, 136, 136, 136, 136, 136, 136, 136 };

	float zero;

	A = (float*)malloc(N*N*sizeof(float));
	A_sub = (float*)malloc(N*N*sizeof(float));
	x = (float*)malloc(N*sizeof(float));
	y = (float*)malloc(N*sizeof(float));

	for (i = 0; i < N; i++){
		for (j = 0; j < N; j++){
			A[i*N + j] = (float)puzzle_sol[i*N + j];

			int sub_i, sub_j;
			sub_i = (i / 4) * 4 + (j / 4);
			sub_j = (i % 4 * 4) + (j % 4);
			A_sub[sub_i*N + sub_j] = (float)puzzle_sol[sub_i*N + sub_j];
		}
	}

	for (i = 0; i < N; i++)
		x[i] = 1.0f;

	for (i = 0; i < N; i++)
		y[i] = 0.0f;


	float* d_A;
	float* d_A_sub;
	float* d_x;
	float* d_y;
	float* d_checksum;

	cudaStat = hipMalloc((void**)&d_A, N*N*sizeof(float));
	cudaStat = hipMalloc((void**)&d_A_sub, N*N*sizeof(float));
	cudaStat = hipMalloc((void**)&d_x, N*sizeof(float));
	cudaStat = hipMalloc((void**)&d_y, N*sizeof(float));
	cudaStat = hipMalloc((void**)&d_checksum, N*sizeof(float));


	stat = hipblasCreate(&handle);
	stat = hipblasSetMatrix(N, N, sizeof(float), A, N, d_A, N);
	stat = hipblasSetMatrix(N, N, sizeof(float), A_sub, N, d_A_sub, N);
	stat = hipblasSetVector(N, sizeof(float), x, 1, d_x, 1);
	stat = hipblasSetVector(N, sizeof(float), y, 1, d_y, 1);
	stat = hipblasSetVector(N, sizeof(float), checksum, 1, d_checksum, 1);

	float alpha = 1.0f;
	float beta = 0.0f;
	float minus = -1.0f;
	//d_y = al*d_a *d_x + bet *d_y
	stat = hipblasSgemv(handle, HIPBLAS_OP_N, N, N, &alpha, d_A, N, d_x, 1, &beta, d_y, 1);//d_y will be sum of the row, 45

	stat = hipblasGetVector(N, sizeof(float), d_y, 1, y, 1);
	/*printf("Sum of each row...\n");
	for (i = 0; i < N; i++){
		printf("%f\n", y[i]);
	}*/
	///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

	//d_y = d_y + minus*d_checksum
	stat = hipblasSaxpy(handle, N, &minus, d_checksum, 1, d_y, 1);//d_y-checksum, 0
	stat = hipblasGetVector(N, sizeof(float), d_y, 1, y, 1);
	/*printf("Difference between correct sum...\n");
	for (i = 0; i < N; i++){
		printf("%f\n", y[i]);
	}*/
	///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	//zero=|d_y[0]|+...+|d_y[9]|
	stat = hipblasSasum(handle, N, d_y, 1, &zero);

	if (zero == 0){
		printf("Row is correct!\n");
	}
	else{
		printf("Row is incorrect\n");
	}
	///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

	//d_y = al*d_a *d_x + bet *d_y
	stat = hipblasSgemv(handle, HIPBLAS_OP_T, N, N, &alpha, d_A, N, d_x, 1, &beta, d_y, 1);//d_y will be sum of the column, 45

	stat = hipblasGetVector(N, sizeof(float), d_y, 1, y, 1);
	/*printf("Sum of each column...\n");
	for (i = 0; i < N; i++){
		printf("%f\n", y[i]);
	}*/
	///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

	//d_y = d_y + minus*d_checksum
	stat = hipblasSaxpy(handle, N, &minus, d_checksum, 1, d_y, 1);//d_y-checksum, 0

	stat = hipblasGetVector(N, sizeof(float), d_y, 1, y, 1);
	/*printf("Difference between correct sum...\n");
	for (i = 0; i < N; i++){
		printf("%f\n", y[i]);
	}*/
	///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

	//zero=|d_y[0]|+...+|d_y[9]|
	stat = hipblasSasum(handle, N, d_y, 1, &zero);

	if (zero == 0){
		printf("Column is correct!\n");
	}
	else{
		printf("Column is incorrect\n");
	}
	///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

	//d_y = al*d_a *d_x + bet *d_y
	stat = hipblasSgemv(handle, HIPBLAS_OP_T, N, N, &alpha, d_A_sub, N, d_x, 1, &beta, d_y, 1);//d_y will be sum of the sub box, 45

	stat = hipblasGetVector(N, sizeof(float), d_y, 1, y, 1);
	/*printf("Sum of each sub box...\n");
	for (i = 0; i < N; i++){
		printf("%f\n", y[i]);
	}*/
	///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

	//d_y = d_y + minus*d_checksum
	stat = hipblasSaxpy(handle, N, &minus, d_checksum, 1, d_y, 1);//d_y-checksum, 0
	stat = hipblasGetVector(N, sizeof(float), d_y, 1, y, 1);
	/*printf("Difference between correct sum...\n");
	for (i = 0; i < N; i++){
		printf("%f\n", y[i]);
	}*/
	///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	//zero=|d_y[0]|+...+|d_y[9]|
	stat = hipblasSasum(handle, N, d_y, 1, &zero);

	if (zero == 0){
		printf("Sub box is correct!\n");
		printf("\nThe output 16x16 sudoku result is correct.\n");
	}
	else{
		printf("Sub box is incorrect\n");
	}

	hipFree(d_A);
	hipFree(d_A_sub);
	hipFree(d_x);
	hipFree(d_y);
	hipFree(d_checksum);
	hipblasDestroy(handle);
	free(A);
	free(x);
	free(y);
	return ;
}

/* 
 * Description:
 *    This function uses backtracking to fill all empty spaces which is not complete with BFS.
 *
 */
__global__ void backtracking(int *new_array, int *empty_pos, int *num_empty, int num_array, int *dev_output){
	unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;

	while(index < num_array){
		
		int empty_index = 0;
		int pos, current_val,val;

		for(empty_index=0; (empty_index < num_empty[index]) && (empty_index >= 0); ){
			/*Get the empty space's position*/
			pos = empty_pos[index*N*N + empty_index];
			
			new_array[index*N*N + pos]++;
			current_val = new_array[index*N*N + pos];
			//printf("%d\n",current_val);
			int r_flag = 1;
			int c_flag = 1;
			int b_flag = 1;
			int a_flag = 1;
			int row = pos/N;
			int col = pos%N;
			/*check row*/
			for(int c = 0; c < N; c++){
				if((row*N+c) != pos){
				  val = new_array[index*N*N + row*N +c];
				  if(val == current_val) r_flag = 0;
				}
			}
			if(r_flag == 1){
				/*check column*/
			    for(int r = 0; r < N; r++){
				   if((r*N+col) != pos){
				       val = new_array[index*N*N + r*N +col];
					   if(val == current_val) c_flag = 0;
			       }
			    }
				
				if(c_flag == 1){
					/*check box*/
				    int r_b = row / n;
                    int c_b = col / n;
				
				    for(int r = 0; r < n; r++){
					   for(int c = 0; c < n; c++){
						   if(( (r_b*n+r)*N + c_b*n + c) != pos){
							   val = new_array[index*N*N + (r_b*n+r)*N + c_b*n + c ];
							   if(val == current_val) b_flag = 0;
						   }
					   }
				    }
					if(b_flag == 1){
						/*check the current value*/
				        if(current_val > 16 ){
                           a_flag = 0;
				           }
					}
				}
			}
				
			if((r_flag == 0) || (c_flag == 0) || (b_flag == 0) || (a_flag == 0)){
			if(current_val >= 16 ){
				/*backtrack to previous attempt*/
				new_array[index*N*N + pos] = 0;
				empty_index--;
			}
			}else{
				empty_index++;
			}	
		}
		
		if(empty_index == num_empty[index]){
		/*copy the result to output array*/
		  for(int i= 0; i < N*N; i++){
		     dev_output[i] = new_array[index*N*N + i];
	      }
		  break;
		}
		index += blockDim.x * gridDim.x; 
	}

	
}

/* 
 * Description:
 *    This function is used to find all possible next ruslts by using breadth
 *  first search.
 *
 */
__global__ void Kernel1(int *pre_array, int *new_array, int num_array, int *counter, int *empty_pos, int *num_empty){
	unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
	
	while(index < num_array){
		
		int emptyflag = 0;
		
		for(int i = index*N*N; i < (index * N * N) + N * N; i++){
			if(pre_array[i] == 0){
				emptyflag = 1;
				int row = (i - index*N*N) / N;
				int col = (i - index*N*N) % N;
				
				/*To check which number could be here*/
				for(int num = 1; num <= N; num++){
					int r_flag = 1;
					int c_flag = 1;
					int b_flag = 1;
					/*check row*/
					for(int c = 0; c < N; c++){
						if(pre_array[index*N*N + row*N + c] == num){
							r_flag = 0;
						}
					}
					if(r_flag == 1){
						/*check column*/
						for(int r = 0; r < N; r++){
							if(pre_array[index*N*N + r*N + col] == num){
								c_flag = 0;
							}
						}
						if(c_flag == 1){
							/*check box*/
							int r_b = row / n;
                            int c_b = col / n;
							for(int r = 0; r < n; r++){
								for(int c = 0; c < n; c++){
									if(pre_array[index*N*N +(r_b*n+r)*N + c_b*n + c] == num){
										b_flag = 0;
									}
								}
							}
							if(b_flag == 1){
								/*this number is available, copy the array*/
								int empty_index = 0;
								int next_index = atomicAdd(counter, 1);
								for(int r = 0; r < N; r++){
									for(int c = 0; c < N; c++){
										new_array[next_index*N*N + r*N + c]=pre_array[index*N*N + r*N + c];
										if(pre_array[index*N*N + r*N + c] == 0 && (r != row || c != col)){
											/*find the position of empty space*/
											empty_pos[next_index*N*N + empty_index] = r*N + c;
											empty_index++;
										}
									}
								}
								new_array[next_index*N*N + row*N + col] = num;
								/*Record the number of empty spaces*/
								num_empty[next_index] = empty_index;
								
							}
						}
					}
				}
			}
			if(emptyflag == 1) break;
		}
		index += blockDim.x * gridDim.x;
	}
	
}


int main(int argc, char* argv[])
{
	FILE *fd,*ff;

	int  *Input,*Output;
	char c='\n';
	if(argc < 5){
		printf("Usage: <number of threads per block> <number of blocks> <input sudoku file> <output file>\n");
		return -1;
	}
	int Blocksize = atoi(argv[1]);
	int NumBlock = atoi(argv[2]);
	
	int *pre_array;  /*Stores the previous version of sudoku boards */
	int *new_array;  /*Stores the new version of sudoku boards */
	int *counter;    /*Total numbers of sudoku boards*/
	int *empty_pos;  /*Stores the position of empty space*/
	int *num_empty;  /*Number of empty space*/
	int *dev_output; /*store the finished version of sudoku board*/
	
	/*maximum number of boards from BFS*/
	int a = pow(2, 26);
	hipMalloc(&pre_array, a * sizeof(int));
	hipMalloc(&new_array, a * sizeof(int));
	hipMalloc(&counter, sizeof(int));
	hipMalloc(&empty_pos, a * sizeof(int));
	hipMalloc(&num_empty, a * sizeof(int));
	hipMalloc(&dev_output, N * N * sizeof(int));
	
	Input = (int*)malloc(N*N*sizeof(int));
    Output = (int*)malloc(N*N*sizeof(int));
    /*Read from input file*/
	fd = fopen(argv[3], "r");
	if(fd == NULL){
		printf("Failed to open file: %s\n",argv[3]);
		return -1;
	}
	
	for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            fscanf(fd, "%d", &Input[i * N + j]);
            //printf("%d\n", Input[i * N + j]);
            //if (!fscanf(a_file, "%c\n", &temp)) {
                //printf("File loading error!\n");
                //return;
            //}

            //if (temp >= '1' && temp <= '9') {
                //board[i * N + j] = (int) (temp - '0');
            //} else {
                //board[i * N + j] = 0;
            //}
        }
    }
	
	/* Initialize */
    hipMemset(counter, 0, sizeof(int));
    hipMemset(new_array, 0, a * sizeof(int));
    hipMemset(pre_array, 0, a * sizeof(int));
	hipMemcpy(pre_array, Input, N*N*sizeof(int), hipMemcpyHostToDevice);
	
    hipEvent_t start1;
    hipEventCreate(&start1);
    hipEvent_t stop1;
    hipEventCreate(&stop1);
    hipEventRecord(start1, NULL);

	int num_array = 1;
	/*number of times to do BFS, and loop_times is an even number here*/
	Kernel1<<<NumBlock, Blocksize>>>(pre_array, new_array, num_array, counter, empty_pos, num_empty);
	hipMemcpy(&num_array, counter, sizeof(int), hipMemcpyDeviceToHost);
	/*Loop to do BFS and to find all emepty position in the borad. Then save all new boards*/	
	for(int i = 0; i < 24; i++){
		hipMemset(counter, 0, sizeof(int));
		if(i % 2==0){
		    Kernel1<<<NumBlock, Blocksize>>>(new_array, pre_array, num_array, counter, empty_pos, num_empty);
		}else{
		    Kernel1<<<NumBlock, Blocksize>>>(pre_array, new_array, num_array, counter, empty_pos, num_empty);
	    }
		hipMemcpy(&num_array, counter, sizeof(int), hipMemcpyDeviceToHost);
		printf("Number of boards created after an iteration %d: %d\n", i, num_array);
	}
	/*Backtracking to complete the board*/
	backtracking<<<NumBlock, Blocksize>>>(new_array, empty_pos, num_empty, num_array, dev_output);
	
	hipEventRecord(stop1, NULL);
    hipEventSynchronize(stop1);
    float msecTotal1 = 0.0f;
    hipEventElapsedTime(&msecTotal1, start1, stop1);
	printf("\n***********************************");
    printf("\nThe execution time is %f ms",msecTotal1);
	printf("\n***********************************\n");
	
	/*print the results*/
	hipMemcpy(Output, dev_output, N * N * sizeof(int), hipMemcpyDeviceToHost);
	
	ff = fopen(argv[4],"w");
	if(ff == NULL){
		printf("Failed to open file: %s\n",argv[4]);
		return -1;
	}
	
	for(int i = 0; i < N; i++){
		for(int j = 0; j < N; j++){
			if(fprintf(ff,"%d\t",Output[N*i+j]) == -1){
				printf("Failed to print\n");
				return -1;
			}
		}
		if(fprintf(ff,"%c",c) == -1){
			printf("Failed to print\n");
			return -1;
		}
	}
	printf("The result has been written into %s\n",argv[4]);
	
	
	for (int i = 0; i < N; i++) {
        if (i % n == 0) {
            printf("--------------------------------------------------\n");
        }

        for (int m = 0; m < N; m++) {
            if (m % n == 0) {
               printf("| ");
            }
            printf("%d ", Output[i * N + m]);
        }

        printf("|\n");
    }
    printf("--------------------------------------------------\n");
	printf("\n\nNow start to check the result\n");
    printf("------------------------------------------------------\n\n");
    /*check the result*/
	check(Output);
    /*free memory*/
	free(Input);
	free(Output);
	hipFree(pre_array);
	hipFree(new_array);
	hipFree(empty_pos);
	hipFree(num_empty);
	hipFree(counter);
	hipFree(dev_output);
	
	
	return 0;
}