#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>


#define N 9
#define n 3


__global__ void backtracking(int *new_array, int *empty_pos, int *num_empty, int num_array, int *dev_output){
	unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;

	while(index < num_array){
		
		int empty_index = 0;
		int pos, current_val,val;

		for(empty_index=0; (empty_index < num_empty[index]) && (empty_index >= 0); ){
			pos = empty_pos[index*N*N + empty_index];
			new_array[index*N*N + pos]++;
			current_val = new_array[index*N*N + pos];
			//printf("%d\n",current_val);
			int r_flag = 1;
			int c_flag = 1;
			int b_flag = 1;
			int a_flag = 1;
			int row = pos/N;
			int col = pos%N;
			for(int c = 0; c < N; c++){
				if((row*N+c) != pos){
				  val = new_array[index*N*N + row*N +c];
				  if(val == current_val) r_flag = 0;
				}
			}
			if(r_flag == 1){
			    for(int r = 0; r < N; r++){
				   if((r*N+col) != pos){
				       val = new_array[index*N*N + r*N +col];
					   if(val == current_val) c_flag = 0;
			       }
			    }
				
				if(c_flag == 1){
				    int ridx = row / n;
                    int cidx = col / n;
				
				    for(int r = 0; r < n; r++){
					   for(int c = 0; c < n; c++){
						   if(( (ridx*n+r)*N + cidx*n + c) != pos){
							   val = new_array[index*N*N + (ridx*n+r)*N + cidx*n + c ];
							   if(val == current_val) b_flag = 0;
						   }
					   }
				    }
					if(b_flag == 1){
				        if(current_val > 9 ){
                           a_flag = 0;
				           }
					}
				}
			}
					
				
			
			if((r_flag == 0) || (c_flag == 0) || (b_flag == 0) || (a_flag == 0)){
			if(current_val >= 9 ){
				new_array[index*N*N + pos] = 0;
				empty_index--;
			}
			}else{
				empty_index++;
			}	
		}
		
		if(empty_index == num_empty[index]){
		
		  for(int i= 0; i < N*N; i++){
		     dev_output[i] = new_array[index*N*N + i];
	      }
		  break;
		}
		index += blockDim.x * gridDim.x; 
	}

	
}



__global__ void Kernel1(int *pre_array, int *new_array, int num_array, int *counter, int *empty_pos, int *num_empty){
	unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
	
	while(index < num_array){
		
		int emptyflag = 0;
		
		for(int i = index*N*N; i < (index * N * N) + N * N; i++){
			if(pre_array[i] == 0){
				emptyflag = 1;
				int row = (i - index*N*N) / N;
				int col = (i - index*N*N) % N;
				
				/*To check which number could be here*/
				for(int num = 1; num <= N; num++){
					int r_flag = 1;
					int c_flag = 1;
					int b_flag = 1;
					/*check row*/
					for(int c = 0; c < N; c++){
						if(pre_array[index*N*N + row*N + c] == num){
							r_flag = 0;
						}
					}
					if(r_flag == 1){
						for(int r = 0; r < N; r++){
							if(pre_array[index*N*N + r*N + col] == num){
								c_flag = 0;
							}
						}
						if(c_flag == 1){
							int r_b = row / n;
                            int c_b = col / n;
							for(int r = 0; r < n; r++){
								for(int c = 0; c < n; c++){
									if(pre_array[index*N*N +(r_b*n+r)*N + c_b*n + c] == num){
										b_flag = 0;
									}
								}
							}
							if(b_flag == 1){
								/*this number is available, copy the array*/
								int empty_index = 0;
								int next_index = atomicAdd(counter, 1);
								for(int r = 0; r < N; r++){
									for(int c = 0; c < N; c++){
										new_array[next_index*N*N + r*N + c]=pre_array[index*N*N + r*N + c];
										if(pre_array[index*N*N + r*N + c] == 0 && (r != row || c != col)){
											empty_pos[next_index*N*N + empty_index] = r*N + c;
											empty_index++;
										}
									}
								}
								new_array[next_index*N*N + row*N + col] = num;
								num_empty[next_index] = empty_index;
								
							}
						}
					}
				}
			}
			if(emptyflag == 1) break;
		}
		index += blockDim.x * gridDim.x;
	}
	
}


int main(int argc, char* argv[])
{
	FILE *fd,*ff;
	char temp;
	int  *Input,*Output;
	char c='\n';
	if(argc < 5){
		printf("Usage: <number of threads per block> <number of blocks> <input sudoku file> <output file>\n");
		return -1;
	}
	int Blocksize = atoi(argv[1]);
	int NumBlock = atoi(argv[2]);
	
	int *pre_array;  /*Stores the previous version of sudoku boards */
	int *new_array;  /*Stores the new version of sudoku boards */
	int *counter;    /*Total numbers of sudoku boards*/
	int *empty_pos;  /*Stores the position of empty space*/
	int *num_empty;  /*Number of empty space*/
	int *dev_output; /*store the finished version of sudoku board*/
	
	int a = pow(2, 26);
	hipMalloc(&pre_array, a * sizeof(int));
	hipMalloc(&new_array, a * sizeof(int));
	hipMalloc(&counter, sizeof(int));
	hipMalloc(&empty_pos, a * sizeof(int));
	hipMalloc(&num_empty, a * sizeof(int));
	hipMalloc(&dev_output, N * N * sizeof(int));
	
	Input = (int*)malloc(N*N*sizeof(int));
    Output = (int*)malloc(N*N*sizeof(int));
    /*Read from input file*/
	fd = fopen(argv[3], "r");
	if(fd == NULL){
		printf("Failed to open file: %s\n",argv[3]);
		return -1;
	}
	
	for(int i = 0; i < N; i++){
		for(int j = 0; j < N; j++){
			if(fscanf(fd,"%c\n",&temp) == -1){
				printf("Failed to read file\n");
				return -1;
			}
			if(temp < '0' || temp > '9'){
				printf("ERROR: Input file is wrong\n");
			}else{
				Input[N*i+j] = (int)(temp - '0');
			}
		}
	}
	
    hipMemset(counter, 0, sizeof(int));
    hipMemset(new_array, 0, a * sizeof(int));
    hipMemset(pre_array, 0, a * sizeof(int));
	hipMemcpy(pre_array, Input, N*N*sizeof(int), hipMemcpyHostToDevice);
	
    hipEvent_t start1;
    hipEventCreate(&start1);
    hipEvent_t stop1;
    hipEventCreate(&stop1);
    hipEventRecord(start1, NULL);

	int num_array = 1;
	Kernel1<<<NumBlock, Blocksize>>>(pre_array, new_array, num_array, counter, empty_pos, num_empty);
	hipMemcpy(&num_array, counter, sizeof(int), hipMemcpyDeviceToHost);
	/*Loop to find all emepty position in the borad and save all new boards*/	
	for(int i = 0; i < 24; i++){
		hipMemset(counter, 0, sizeof(int));
		if(i % 2==0){
		    Kernel1<<<NumBlock, Blocksize>>>(new_array, pre_array, num_array, counter, empty_pos, num_empty);
		}else{
		    Kernel1<<<NumBlock, Blocksize>>>(pre_array, new_array, num_array, counter, empty_pos, num_empty);
	    }
		hipMemcpy(&num_array, counter, sizeof(int), hipMemcpyDeviceToHost);
		printf("Number of boards created after an iteration %d: %d\n", i, num_array);
	}
	
	backtracking<<<NumBlock, Blocksize>>>(new_array, empty_pos, num_empty, num_array, dev_output);
	
	hipEventRecord(stop1, NULL);
    hipEventSynchronize(stop1);
    float msecTotal1 = 0.0f;
    hipEventElapsedTime(&msecTotal1, start1, stop1);
	printf("\n***********************************");
    printf("\nThe execution time is %f ms",msecTotal1);
	printf("\n***********************************\n");
	/*print the results*/
	
	hipMemcpy(Output, dev_output, N * N * sizeof(int), hipMemcpyDeviceToHost);
	
	ff = fopen(argv[4],"w");
	if(ff == NULL){
		printf("Failed to open file: %s\n",argv[4]);
		return -1;
	}
	
	for(int i = 0; i < N; i++){
		for(int j = 0; j < N; j++){
			if(fprintf(ff,"%d\t",Output[N*i+j]) == -1){
				printf("Failed to print\n");
				return -1;
			}
		}
		if(fprintf(ff,"%c",c) == -1){
			printf("Failed to print\n");
			return -1;
		}
	}
	printf("The result has been written into %s\n",argv[4]);
	
	
	for (int i = 0; i < N; i++) {
        if (i % n == 0) {
            printf("-------------------------\n");
        }

        for (int m = 0; m < N; m++) {
            if (m % n == 0) {
               printf("| ");
            }
            printf("%d ", Output[i * N + m]);
        }

        printf("|\n");
    }
    printf("-------------------------\n");
	
	free(Input);
	free(Output);
	hipFree(pre_array);
	hipFree(new_array);
	hipFree(empty_pos);
	hipFree(num_empty);
	hipFree(counter);
	hipFree(dev_output);
	
	
	return 0;
}